#include "hip/hip_runtime.h"
#define GET_PROF

#include "hip/hip_runtime.h"
#include "include/helper/cuda/cublas_error_check.h"
#include "include/helper/cuda/cusparse_error_check.h"
#include <assert.h>
#include <stdio.h>

#include "basic_operations.hpp"
#include "dataStructures/hd_data.hpp"
#include "dataStructures/matrix_element.hpp"
#include "helper/cuda/cuda_error_check.h"
#include "helper/cuda/cuda_reduction_operation.hpp"
#include "helper/cuda/cuda_thread_manager.hpp"

ChronoProfiler profDot;
void PrintDotProfiler() { profDot.print(); }

hipsparseHandle_t cusparseHandle = NULL;
hipblasHandle_t cublasHandle = NULL;

void dot(d_spmatrix &d_mat, d_vector &x, d_vector &result, bool synchronize) {
    if (!cusparseHandle)
        cusparseErrchk(hipsparseCreate(&cusparseHandle));
    assert(d_mat.isDevice && x.isDevice && result.isDevice);
    if (&x == &result) {
        printf("Error: X and Result vectors should not be the same instance\n");
        return;
    }
    T one = 1.0;
    T zero = 0.0;
    size_t size = 0;
    T *buffer;
    auto mat_descr = d_mat.MakeSpDescriptor();
    auto x_descr = x.MakeDescriptor();
    auto res_descr = result.MakeDescriptor();
    cusparseErrchk(hipsparseSpMV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, mat_descr,
        x_descr, &zero, res_descr, T_Cuda, HIPSPARSE_MV_ALG_DEFAULT, &size));
    if (size > 0)
        printf("Alert! size >0 \n");
    hipMalloc(&buffer, size);
    cusparseErrchk(hipsparseSpMV(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, mat_descr,
        x_descr, &zero, res_descr, T_Cuda, HIPSPARSE_MV_ALG_DEFAULT, buffer));
    hipsparseDestroyDnVec(x_descr);
    hipsparseDestroyDnVec(res_descr);
    hipsparseDestroySpMat(mat_descr);
}

d_vector buffer(0);

__global__ void DotK(d_vector &x, d_vector &y, d_vector &buffer) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= x.n)
        return;
    buffer.data[i] = x.data[i] * y.data[i];
    return;
}

void dot(d_vector &x, d_vector &y, T &result, bool synchronize) {
    assert(x.isDevice && y.isDevice);
    assert(x.n == y.n);

    if (!cublasHandle)
        cublasErrchk(hipblasCreate(&cublasHandle));

#ifdef USE_DOUBLE
    cublasErrchk(hipblasDdot(cublasHandle, x.n, x.data, 1, y.data, 1, &result));
#else
    cublasErrchk(hipblasSdot(cublasHandle, x.n, x.data, sizeof(T), y.data,
                            sizeof(T), &result));
#endif
    // dim3Pair threadblock = Make1DThreadBlock(x.n);
    // if (buffer.n < x.n)
    //     buffer.Resize(x.n);
    // else
    //     buffer.n = x.n;

    // DotK<<<threadblock.block, threadblock.thread>>>(*(d_vector *)x._device,
    //                                                 *(d_vector *)y._device,
    //                                                 *(d_vector
    //                                                 *)buffer._device);
    // ReductionOperation(buffer, sum);
    // hipMemcpy(&result, buffer.data, sizeof(T), hipMemcpyDeviceToDevice);
    if (synchronize) {
        gpuErrchk(hipDeviceSynchronize());
    } else
        return;
}

__global__ void VectorSumK(d_vector &a, d_vector &b, T &alpha, d_vector &c) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= a.n)
        return;
    c.data[i] = a.data[i] + b.data[i] * alpha;
};

void vector_sum(d_vector &a, d_vector &b, T &alpha, d_vector &c,
                bool synchronize) {
    assert(a.isDevice && b.isDevice);
    assert(a.n == b.n);
    dim3Pair threadblock = Make1DThreadBlock(a.n);
    VectorSumK<<<threadblock.block, threadblock.thread>>>(
        *(d_vector *)a._device, *(d_vector *)b._device, alpha,
        *(d_vector *)c._device);
    if (synchronize)
        gpuErrchk(hipDeviceSynchronize());
}

void vector_sum(d_vector &a, d_vector &b, d_vector &c, bool synchronize) {
    hd_data<T> alpha(1.0);
    vector_sum(a, b, alpha(true), c, synchronize);
}

__device__ inline bool IsSup(MatrixElement &it_a, MatrixElement &it_b) {
    return (it_a.i == it_b.i && it_a.j > it_b.j) || it_a.i > it_b.i;
};

__device__ inline bool IsEqu(MatrixElement &it_a, MatrixElement &it_b) {
    return (it_a.i == it_b.i && it_a.j == it_b.j);
};

__device__ inline bool IsSupEqu(MatrixElement &it_a, MatrixElement &it_b) {
    return (it_a.i == it_b.i && it_a.j >= it_b.j) || it_a.i > it_b.i;
};

__global__ void SumNNZK(d_spmatrix &a, d_spmatrix &b, int *nnz) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= a.rows)
        return;
    if (i == 0)
        nnz[0] = 0;
    MatrixElement it_a(a.rowPtr[i], &a);
    MatrixElement it_b(b.rowPtr[i], &b);
    nnz[i + 1] = 0;
    while (it_a.i == i || it_b.i == i) {
        if (IsEqu(it_a, it_b)) {
            it_a.Next();
            it_b.Next();
            nnz[i + 1] += 1;
        } else if (IsSup(it_a, it_b)) {
            it_b.Next();
            nnz[i + 1] += 1;
        } else if (IsSup(it_b, it_a)) {
            it_a.Next();
            nnz[i + 1] += 1;
        } else {
            printf("Error! Nobody was iterated in SumNNZK function.\n");
            return;
        }
    }
    return;
}

__global__ void SetValuesK(d_spmatrix &a, d_spmatrix &b, T &alpha,
                           d_spmatrix &c) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= c.rows)
        return;
    MatrixElement it_a(a.rowPtr[i], &a);
    MatrixElement it_b(b.rowPtr[i], &b);
    int k = c.rowPtr[i];
    if (k >= c.nnz) {
        printf("Error! In matrix sum, at %i\n", i);
        return;
    }
    while (it_a.i == i || it_b.i == i) {
        if (IsEqu(it_a, it_b)) {
            c.colPtr[k] = it_a.j;
            c.data[k] = it_a.val[0] + alpha * it_b.val[0];
            it_a.Next();
            it_b.Next();
        } else if (IsSup(it_a, it_b)) {
            c.colPtr[k] = it_b.j;
            c.data[k] = alpha * it_b.val[0];
            it_b.Next();
        } else if (IsSup(it_b, it_a)) {
            c.colPtr[k] = it_a.j;
            c.data[k] = it_a.val[0];
            it_a.Next();
        } else {
            printf("Error! Nobody was iterated in SumNNZK function.\n");
            return;
        }
        k++;
    }
    return;
}

void matrix_sum(d_spmatrix &a, d_spmatrix &b, T &alpha, d_spmatrix &c) {
    // This method is only impleted in the specific case of CSR matrices
    assert(a.type == CSR && b.type == CSR);
    assert(a.rows == b.rows && a.cols == b.cols);
    c.rows = 1 * a.rows;
    c.cols = 1 * a.cols;
    c.type = CSR;
    int *nnzs;
    hipMalloc(&nnzs, sizeof(int) * (a.rows + 1));
    auto tb = Make1DThreadBlock(a.rows);
    SumNNZK<<<tb.block, tb.thread>>>(*a._device, *b._device, nnzs);
    ReductionIncreasing(nnzs, a.rows + 1);
    hd_data<int> nnz(&nnzs[a.rows], true);
    c.SetNNZ(nnz());

    gpuErrchk(hipMemcpy(c.rowPtr, nnzs, sizeof(int) * (a.rows + 1),
                         hipMemcpyDeviceToDevice));

    SetValuesK<<<tb.block, tb.thread>>>(*a._device, *b._device, alpha,
                                        *c._device);
    gpuErrchk(hipDeviceSynchronize());
    return;
}

void matrix_sum(d_spmatrix &a, d_spmatrix &b, d_spmatrix &c) {
    hd_data<T> d_alpha(1.0);
    matrix_sum(a, b, d_alpha(true), c);
}

__global__ void ScalarMultK(T *data, int n, T &alpha) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= n)
        return;
    data[i] *= alpha;
    return;
}

void scalar_mult(d_spmatrix &a, T &alpha) {
    assert(a.isDevice);
    dim3Pair threadblock = Make1DThreadBlock(a.nnz);
    ScalarMultK<<<threadblock.block, threadblock.thread>>>(a.data, a.nnz,
                                                           alpha);
}
void scalar_mult(d_vector &a, T &alpha) {
    assert(a.isDevice);
    dim3Pair threadblock = Make1DThreadBlock(a.n);
    ScalarMultK<<<threadblock.block, threadblock.thread>>>(a.data, a.n, alpha);
}
