#include "hip/hip_runtime.h"
#define GET_PROF

#include "hip/hip_runtime.h"
#include "include/hediHelper/cuda/cusparse_error_check.h"
#include <assert.h>
#include <stdio.h>

#include "basic_operations.hpp"
#include "dataStructures/hd_data.hpp"
#include "dataStructures/matrix_element.hpp"
#include "hediHelper/cuda/cuda_error_check.h"
#include "hediHelper/cuda/cuda_reduction_operation.hpp"
#include "hediHelper/cuda/cuda_thread_manager.hpp"

ChronoProfiler profDot;
void PrintDotProfiler() { profDot.Print(); }

hipsparseHandle_t handle = NULL;

void Dot(D_SparseMatrix &d_mat, D_Array &x, D_Array &result, bool synchronize) {
    profDot.Start("Prep");
    if (!handle)
        cusparseErrchk(hipsparseCreate(&handle));
    assert(d_mat.isDevice && x.isDevice && result.isDevice);
    if (&x == &result) {
        printf("Error: X and Result vectors should not be the same instance\n");
        return;
    }
    profDot.Start("Alloc");
    T one = 1.0;
    T zero = 0.0;
    size_t size = 0;
    T *buffer;
    profDot.Start("BuffSize");
    auto mat_descr = d_mat.MakeSpDescriptor();
    auto x_descr = x.MakeDescriptor();
    auto res_descr = result.MakeDescriptor();
    cusparseErrchk(hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, mat_descr, x_descr,
        &zero, res_descr, T_Cuda, HIPSPARSE_MV_ALG_DEFAULT, &size));
    if (size > 0)
        printf("Alert! Size >0 \n");
    hipMalloc(&buffer, size);
    profDot.Start("Computation");
    cusparseErrchk(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one,
                                mat_descr, x_descr, &zero, res_descr, T_Cuda,
                                HIPSPARSE_MV_ALG_DEFAULT, buffer));
    hipsparseDestroyDnVec(x_descr);
    hipsparseDestroyDnVec(res_descr);
    hipsparseDestroySpMat(mat_descr);
    profDot.End();
}

D_Array buffer(0);

__global__ void DotK(D_Array &x, D_Array &y, D_Array &buffer) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= x.n)
        return;
    buffer.vals[i] = x.vals[i] * y.vals[i];
    return;
}

void Dot(D_Array &x, D_Array &y, T &result, bool synchronize) {
    assert(x.isDevice && y.isDevice);
    assert(x.n == y.n);
    dim3Pair threadblock = Make1DThreadBlock(x.n);
    if (buffer.n < x.n)
        buffer.Resize(x.n);
    else
        buffer.n = x.n;

    DotK<<<threadblock.block, threadblock.thread>>>(*(D_Array *)x._device,
                                                    *(D_Array *)y._device,
                                                    *(D_Array *)buffer._device);
    ReductionOperation(buffer, sum);
    hipMemcpy(&result, buffer.vals, sizeof(T), hipMemcpyDeviceToDevice);
    if (synchronize) {
        gpuErrchk(hipDeviceSynchronize());
    } else
        return;
}

__global__ void VectorSumK(D_Array &a, D_Array &b, T &alpha, D_Array &c) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= a.n)
        return;
    c.vals[i] = a.vals[i] + b.vals[i] * alpha;
};

void VectorSum(D_Array &a, D_Array &b, T &alpha, D_Array &c, bool synchronize) {
    assert(a.isDevice && b.isDevice);
    assert(a.n == b.n);
    dim3Pair threadblock = Make1DThreadBlock(a.n);
    VectorSumK<<<threadblock.block, threadblock.thread>>>(
        *(D_Array *)a._device, *(D_Array *)b._device, alpha,
        *(D_Array *)c._device);
    if (synchronize)
        gpuErrchk(hipDeviceSynchronize());
}

void VectorSum(D_Array &a, D_Array &b, D_Array &c, bool synchronize) {
    HDData<T> alpha(1.0);
    VectorSum(a, b, alpha(true), c, synchronize);
}

__device__ inline bool IsSup(MatrixElement &it_a, MatrixElement &it_b) {
    return (it_a.i == it_b.i && it_a.j > it_b.j) || it_a.i > it_b.i;
};

__device__ inline bool IsEqu(MatrixElement &it_a, MatrixElement &it_b) {
    return (it_a.i == it_b.i && it_a.j == it_b.j);
};

__device__ inline bool IsSupEqu(MatrixElement &it_a, MatrixElement &it_b) {
    return (it_a.i == it_b.i && it_a.j >= it_b.j) || it_a.i > it_b.i;
};

__global__ void SumNNZK(D_SparseMatrix &a, D_SparseMatrix &b, int *nnz) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= a.rows)
        return;
    if (i == 0)
        nnz[0] = 0;
    MatrixElement it_a(a.rowPtr[i], &a);
    MatrixElement it_b(b.rowPtr[i], &b);
    nnz[i + 1] = 0;
    while (it_a.i == i || it_b.i == i) {
        if (IsEqu(it_a, it_b)) {
            it_a.Next();
            it_b.Next();
            nnz[i + 1] += 1;
        } else if (IsSup(it_a, it_b)) {
            it_b.Next();
            nnz[i + 1] += 1;
        } else if (IsSup(it_b, it_a)) {
            it_a.Next();
            nnz[i + 1] += 1;
        } else {
            printf("Error! Nobody was iterated in SumNNZK function.\n");
            return;
        }
    }
    return;
}

__global__ void SetValuesK(D_SparseMatrix &a, D_SparseMatrix &b, T &alpha,
                           D_SparseMatrix &c) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= c.rows)
        return;
    MatrixElement it_a(a.rowPtr[i], &a);
    MatrixElement it_b(b.rowPtr[i], &b);
    int k = c.rowPtr[i];
    if (k >= c.nnz) {
        printf("Error! In matrix sum, at %i\n", i);
        return;
    }
    while (it_a.i == i || it_b.i == i) {
        if (IsEqu(it_a, it_b)) {
            c.colPtr[k] = it_a.j;
            c.vals[k] = it_a.val[0] + alpha * it_b.val[0];
            it_a.Next();
            it_b.Next();
        } else if (IsSup(it_a, it_b)) {
            c.colPtr[k] = it_b.j;
            c.vals[k] = alpha * it_b.val[0];
            it_b.Next();
        } else if (IsSup(it_b, it_a)) {
            c.colPtr[k] = it_a.j;
            c.vals[k] = it_a.val[0];
            it_a.Next();
        } else {
            printf("Error! Nobody was iterated in SumNNZK function.\n");
            return;
        }
        k++;
    }
    return;
}

void MatrixSum(D_SparseMatrix &a, D_SparseMatrix &b, T &alpha,
               D_SparseMatrix &c) {
    // This method is only impleted in the specific case of CSR matrices
    assert(a.type == CSR && b.type == CSR);
    assert(a.rows == b.rows && a.cols == b.cols);
    c.rows = 1 * a.rows;
    c.cols = 1 * a.cols;
    c.type = CSR;
    int *nnzs;
    hipMalloc(&nnzs, sizeof(int) * (a.rows + 1));
    auto tb = Make1DThreadBlock(a.rows);
    SumNNZK<<<tb.block, tb.thread>>>(*a._device, *b._device, nnzs);
    ReductionIncreasing(nnzs, a.rows + 1);
    HDData<int> nnz(&nnzs[a.rows], true);
    c.SetNNZ(nnz());

    gpuErrchk(hipMemcpy(c.rowPtr, nnzs, sizeof(int) * (a.rows + 1),
                         hipMemcpyDeviceToDevice));

    SetValuesK<<<tb.block, tb.thread>>>(*a._device, *b._device, alpha,
                                        *c._device);
    gpuErrchk(hipDeviceSynchronize());
    return;
}

void MatrixSum(D_SparseMatrix &a, D_SparseMatrix &b, D_SparseMatrix &c) {
    HDData<T> d_alpha(1.0);
    MatrixSum(a, b, d_alpha(true), c);
}

__global__ void ScalarMultK(T *vals, int n, T &alpha) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= n)
        return;
    vals[i] *= alpha;
    return;
}

void ScalarMult(D_SparseMatrix &a, T &alpha) {
    assert(a.isDevice);
    dim3Pair threadblock = Make1DThreadBlock(a.nnz);
    ScalarMultK<<<threadblock.block, threadblock.thread>>>(a.vals, a.nnz,
                                                           alpha);
}
void ScalarMult(D_Array &a, T &alpha) {
    assert(a.isDevice);
    dim3Pair threadblock = Make1DThreadBlock(a.n);
    ScalarMultK<<<threadblock.block, threadblock.thread>>>(a.vals, a.n, alpha);
}
