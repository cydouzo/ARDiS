#include "hip/hip_runtime.h"
#include "dataStructures/helper/apply_operation.h"
#include "reaction_computer.h"
#include "system.hpp"

System::System(int size) : state(size), solver(size), b(size){};

void System::AddReaction(std::string reag, int kr, std::string prod, int kp,
                         T rate) {
    std::vector<stochCoeff> input;
    std::vector<stochCoeff> output;
    input.push_back(std::pair<std::string, int>(reag, kr));
    output.push_back(std::pair<std::string, int>(prod, kp));
    AddReaction(input, output, rate);
}
void System::AddReaction(std::vector<stochCoeff> input,
                         std::vector<stochCoeff> output, T factor) {
    AddReaction(Reaction(input, output, factor));
}
void System::AddReaction(Reaction reaction) {
    for (auto species : std::get<0>(reaction)) {
        if (state.names.find(species.first) == state.names.end()) {
            std::cout << "\"" << species.first << "\""
                      << "\n";
            throw std::invalid_argument("^ This species is invalid\n");
        }
    }
    for (auto species : std::get<1>(reaction)) {
        if (state.names.find(species.first) == state.names.end()) {
            std::cout << "\"" << species.first << "\""
                      << "\n";
            throw std::invalid_argument("^ This species is invalid\n");
        }
    }
    reactions.push_back(reaction);
};

void System::LoadDampnessMatrix(D_SparseMatrix &damp_mat) {
    this->damp_mat = &damp_mat;
}
void System::LoadStiffnessMatrix(D_SparseMatrix &stiff_mat) {
    this->stiff_mat = &stiff_mat;
}

__global__ void PruneK(D_Array **state, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= state[0]->n)
        return;
    for (int k = 0; k < size; k++) {
        if (state[k]->vals[i] < 0)
            state[k]->vals[i] = 0;
    }
}

void System::Prune(T value) {
    auto tb = Make1DThreadBlock(state.size);
    for (auto &vect : state.data)
        vect->Prune(value);
}

const T drain = 1.e-15;

void System::IterateReaction(T dt, bool degradation) {
    profiler.Start("Reaction");
    auto drainLambda = [] __device__(T & x) { x -= drain; };
    for (auto species : state.data) {
        ApplyFunction(*species, drainLambda);
        species->Prune();
    }
    for (auto reaction : reactions) {
        ConsumeReaction(state, reaction, std::get<2>(reaction) * dt);
    }
    profiler.End();
}

int i = 0;
bool System::IterateDiffusion(T dt) {
    profiler.Start("Diffusion Initialization");
    if (damp_mat == nullptr || stiff_mat == nullptr) {
        printf("Error! Stiffness and Dampness matrices not loaded\n");
        return false;
    }
    profiler.End();
    if (last_used_dt != dt) {
        printf("Building a diffusion matrix for dt = %f ... ", dt);
        HDData<T> m(-dt);
        MatrixSum(*damp_mat, *stiff_mat, m(true), diffusion_matrix);
        printf("Done!\n");
        last_used_dt = dt;
    }
    profiler.Start("Diffusion");
    for (auto &species : state.data) {
        Dot(*damp_mat, *species, b);
        if (!solver.CGSolve(diffusion_matrix, b, *species, epsilon)) {
            printf("Warning: It did not converge on %i\n", i);
            species->Print(20);
            return false;
        }
    }
    i++;
    return true;
    profiler.End();
}

void System::Print(int printCount) {
    state.Print(printCount);
    for (int i = 0; i < reactions.size(); i++) {
        for (auto coeff : std::get<0>(reactions.at(i)))
            std::cout << coeff.second << "." << coeff.first << " + ";
        std::cout << "-> ";
        for (auto coeff : std::get<1>(reactions.at(i)))
            std::cout << coeff.second << "." << coeff.first << " + ";
        std::cout << "k=" << std::get<2>(reactions.at(i)) << "\n";
    }
    std::cout << "Global Profiler : \n";
    profiler.Print();
    std::cout << "Operation Profiler : \n";
    solver.profiler.Print();
}

void System::SetEpsilon(T epsilon) { this->epsilon = epsilon; }

System::~System(){
    // if (damp_mat != nullptr)
    //     delete damp_mat;
    // if (stiff_mat != nullptr)
    //     delete stiff_mat;
};
