#include "hip/hip_runtime.h"
#include "dataStructures/helper/apply_operation.h"
#include "reaction_computer.h"
#include "system.hpp"
#include <fstream>

System::System(int size) : state(size), solver(size), b(size){};

void System::AddReaction(std::string reag, int kr, std::string prod, int kp,
                         T rate) {
    std::vector<stochCoeff> input;
    std::vector<stochCoeff> output;
    input.push_back(std::pair<std::string, int>(reag, kr));
    output.push_back(std::pair<std::string, int>(prod, kp));
    AddReaction(input, output, rate);
}
void System::AddReaction(std::vector<stochCoeff> input,
                         std::vector<stochCoeff> output, T factor) {
    AddReaction(Reaction(input, output, factor));
}
void System::AddReaction(Reaction reaction) {
    for (auto species : std::get<0>(reaction)) {
        if (state.names.find(species.first) == state.names.end()) {
            std::cout << "\"" << species.first << "\""
                      << "\n";
            throw std::invalid_argument("^ This species is invalid\n");
        }
    }
    for (auto species : std::get<1>(reaction)) {
        if (state.names.find(species.first) == state.names.end()) {
            std::cout << "\"" << species.first << "\""
                      << "\n";
            throw std::invalid_argument("^ This species is invalid\n");
        }
    }
    reactions.push_back(reaction);
};

void System::LoadDampnessMatrix(D_SparseMatrix &damp_mat) {
    this->damp_mat = &damp_mat;
}
void System::LoadStiffnessMatrix(D_SparseMatrix &stiff_mat) {
    this->stiff_mat = &stiff_mat;
}

__global__ void PruneK(D_Array **state, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= state[0]->n)
        return;
    for (int k = 0; k < size; k++) {
        if (state[k]->vals[i] < 0)
            state[k]->vals[i] = 0;
    }
}

void System::Prune(T value) {
    auto tb = Make1DThreadBlock(state.size);
    for (auto &vect : state.data)
        vect->Prune(value);
}

void System::IterateReaction(T dt, bool degradation) {
#ifndef NDEBUG_PROFILING
    profiler.Start("Reaction");
#endif
    T drainXdt = drain * dt;
    auto drainLambda = [drainXdt] __device__(T & x) { x -= drainXdt; };
    for (auto species : state.data) {
        ApplyFunction(*species, drainLambda);
        species->Prune();
    }
    for (auto reaction : reactions) {
        ConsumeReaction(state, reaction, std::get<2>(reaction) * dt);
    }
#ifndef NDEBUG_PROFILING
    profiler.End();
#endif
}

bool System::IterateDiffusion(T dt) {
#ifndef NDEBUG_PROFILING
    profiler.Start("Diffusion Initialization");
#endif
    if (damp_mat == nullptr || stiff_mat == nullptr) {
        printf("Error! Stiffness and Dampness matrices not loaded\n");
        return false;
    }
    if (last_used_dt != dt) {
        printf("Building a diffusion matrix for dt = %f ... ", dt);
        HDData<T> m(-dt);
        MatrixSum(*damp_mat, *stiff_mat, m(true), diffusion_matrix);
        printf("Done!\n");
        last_used_dt = dt;
    }
#ifndef NDEBUG_PROFILING
    profiler.Start("Diffusion");
#endif
    for (auto &species : state.data) {
        Dot(*damp_mat, *species, b);
        if (!solver.CGSolve(diffusion_matrix, b, *species, epsilon)) {
            printf("Warning: It did not converge at time %f\n", t);
            species->Print(20);
            return false;
        }
    }

#ifndef NDEBUG_PROFILING
    profiler.End();
#endif

    // std::ofstream fout;
    // fout.open("output/CgmIterCount", std::ios_base::app);
    // // std::cout << t << "\t" << solver.n_iter_last << "\n";
    // fout << t << "\t" << solver.n_iter_last << "\n";
    // fout.close();

    t += dt;
    return true;
}

void System::Print(int printCount) {
    state.Print(printCount);
    for (int i = 0; i < reactions.size(); i++) {
        for (auto coeff : std::get<0>(reactions.at(i)))
            std::cout << coeff.second << "." << coeff.first << " + ";
        std::cout << "-> ";
        for (auto coeff : std::get<1>(reactions.at(i)))
            std::cout << coeff.second << "." << coeff.first << " + ";
        std::cout << "k=" << std::get<2>(reactions.at(i)) << "\n";
    }
#ifndef NDEBUG_PROFILING
    std::cout << "Global Profiler : \n";
    profiler.Print();
    std::cout << "Operation Profiler : \n";
    solver.profiler.Print();
#endif
}

void System::SetEpsilon(T epsilon) { this->epsilon = epsilon; }

System::~System(){
    // if (damp_mat != nullptr)
    //     delete damp_mat;
    // if (stiff_mat != nullptr)
    //     delete stiff_mat;
};
