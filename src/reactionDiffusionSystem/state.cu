#include <stdio.h>

#include "state.hpp"

State::State(int size) : size(size) {}

D_Array &State::AddSpecies(std::string name) {
    names[name] = data.size();
    data.push_back(size);
    return data.at(data.size() - 1);
}

D_Array &State::GetSpecies(std::string name) { return data.at(names[name]); }

void State::Print(int i) {
    for (auto name : names) {
        std::cout << name.first << " : ";
        data.at(name.second).Print(i);
    }
}

State::~State() {}

D_Array **State::GetDeviceState() {
    D_Array **output = new D_Array *[data.size()];
    D_Array **d_output;
    hipMalloc(&d_output, sizeof(D_Array *) * data.size());
    for (int i = 0; i < data.size(); i++) {
        output[i] = data.at(i)._device;
    }
    hipMemcpy(d_output, output, sizeof(D_Array *) * data.size(),
               hipMemcpyHostToDevice);
    delete output;
    return d_output;
}

void State::FreeDeviceState(D_Array **freeMe) { hipFree(freeMe); }
