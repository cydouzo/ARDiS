#include <stdio.h>

#include "helper/cuda/cuda_error_check.h"
#include "state.hpp"
state::state(int size) : vector_size(size) {}

d_vector &state::add_species(std::string name) {
    names[name] = n_species();
    vector_holder.push_back(d_vector(vector_size));
    device_data.resize(n_species());
    d_vector *new_device_data[n_species()];
    for (int i = 0; i < n_species(); i++)
        new_device_data[i] = (d_vector *)vector_holder.at(i)._device;
    gpuErrchk(hipMemcpy(device_data.data, new_device_data,
                         sizeof(d_vector *) * n_species(),
                         hipMemcpyHostToDevice));
    return vector_holder.at(n_species() - 1);
}

d_vector &state::get_species(std::string name) {
    auto findRes = names.find(name);
    if (findRes == names.end()) {
        std::cout << "\"" << name << "\"\n";
        throw std::invalid_argument("^ This species is invalid\n");
    }
    return vector_holder.at(names[name]);
}

int state::size() { return vector_size; }
int state::n_species() { return vector_holder.size(); }

void state::print(int i) {
    for (auto name : names) {
        std::cout << name.first << " : ";
        vector_holder.at(name.second).print(i);
    }
}

state::~state() {}
