#include "mesh.hpp"
#include <hip/hip_runtime.h>

D_Mesh::D_Mesh(int n) : X(n), Y(n) {}
D_Mesh::D_Mesh(int n, T *x, T *y) : X(n), Y(n) {}
D_Mesh::D_Mesh(D_Vector &X, D_Vector &Y) : X(X), Y(Y) { assert(X.n == Y.n); }

__host__ __device__ int D_Mesh::size() { return X.n; }

D_Mesh::~D_Mesh() {}