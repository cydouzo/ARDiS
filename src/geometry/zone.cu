#include "hip/hip_runtime.h"
#include "zone.hpp"

SimpleZone SimpleZone::all = SimpleZone(true);
SimpleZone SimpleZone::none = SimpleZone(false);

rect_zone::rect_zone() : rect_zone(0, 0, 0, 0){};
rect_zone::rect_zone(T x0, T y0, T x1, T y1) : x0(x0), x1(x1), y0(y0), y1(y1){};
rect_zone::rect_zone(point2d p0, point2d p1)
    : rect_zone(p0.x, p0.y, p1.x, p1.y){};

__device__ __host__ bool rect_zone::is_inside(T x, T y) {
    return x0 <= x && x1 >= x && y0 <= y && y1 >= y;
}
__device__ __host__ bool rect_zone::is_inside(point2d p) {
    return is_inside(p.x, p.y);
}

tri_zone::tri_zone() : tri_zone(0, 0, 0, 0, 0, 0){};
tri_zone::tri_zone(T x0, T y0, T x1, T y1, T x2, T y2)
    : x0(x0), x1(x1), y0(y0), y1(y1), x2(x2), y2(y2){};
tri_zone::tri_zone(point2d p0, point2d p1, point2d p2)
    : tri_zone(p0.x, p0.y, p1.x, p1.y, p2.x, p2.y){};

__device__ __host__ T Sign(T x0, T y0, T x1, T y1, T x2, T y2) {
    return (x0 - x2) * (y1 - y2) - (x1 - x2) * (y0 - y2);
}

__device__ __host__ bool tri_zone::is_inside(T x, T y) {
    bool b1 = Sign(x, y, x0, y0, x1, y1) < 0.0;
    bool b2 = Sign(x, y, x1, y1, x2, y2) < 0.0;
    bool b3 = Sign(x, y, x2, y2, x0, y0) < 0.0;
    return ((b1 == b2) && (b2 == b3));
}

__device__ __host__ bool tri_zone::is_inside(point2d p) {
    return is_inside(p.x, p.y);
}
