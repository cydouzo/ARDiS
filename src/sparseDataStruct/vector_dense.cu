#include "sparseDataStruct/helper/vector_helper.h"
#include "sparseDataStruct/vector_dense.hpp"

__host__ VectorDense::VectorDense(int n, bool isDevice)
    : n(n), isDevice(isDevice) {
    MemAlloc();
}

__host__ VectorDense::VectorDense(const VectorDense &m, bool copyToOtherMem)
    : VectorDense(m.n, m.isDevice ^ copyToOtherMem) {
    hipMemcpyKind memCpy =
        (m.isDevice)
            ? (isDevice) ? hipMemcpyDeviceToDevice : hipMemcpyDeviceToHost
            : (isDevice) ? hipMemcpyHostToDevice : hipMemcpyHostToHost;
    gpuErrchk(hipMemcpy(vals, m.vals, sizeof(T) * n, memCpy));
}

__host__ void VectorDense::MemAlloc() {
    if (isDevice) {
        gpuErrchk(hipMalloc(&vals, n * sizeof(T)));
        gpuErrchk(hipMalloc(&_device, sizeof(VectorDense)));
        gpuErrchk(hipMemcpy(_device, this, sizeof(VectorDense),
                             hipMemcpyHostToDevice));
    } else {
        vals = new T[n];
    }
}

__host__ __device__ void VectorDense::Print() {
    printf("[ ");
#ifndef __CUDA_ARCH__
    if (isDevice) {
        printVector<<<1, 1>>>(_device);
        hipDeviceSynchronize();
    } else
#endif
        printVectorBody(this);
}

__host__ VectorDense::~VectorDense() {
    if (isDevice) {
        hipFree(vals);
        hipFree(_device);
    } else {
        delete[] vals;
    }
}
