#include "sparseDataStruct/helper/vector_helper.h"
#include "sparseDataStruct/vector_dense.hpp"

__host__ D_Array::D_Array(int n, bool isDevice)
    : n(n), isDevice(isDevice) {
    MemAlloc();
}

__host__ D_Array::D_Array(const D_Array &m, bool copyToOtherMem)
    : D_Array(m.n, m.isDevice ^ copyToOtherMem) {
    hipMemcpyKind memCpy =
        (m.isDevice)
            ? (isDevice) ? hipMemcpyDeviceToDevice : hipMemcpyDeviceToHost
            : (isDevice) ? hipMemcpyHostToDevice : hipMemcpyHostToHost;
    gpuErrchk(hipMemcpy(vals, m.vals, sizeof(T) * n, memCpy));
}

__host__ void D_Array::MemAlloc() {
    if (isDevice) {
        gpuErrchk(hipMalloc(&vals, n * sizeof(T)));
        gpuErrchk(hipMalloc(&_device, sizeof(D_Array)));
        gpuErrchk(hipMemcpy(_device, this, sizeof(D_Array),
                             hipMemcpyHostToDevice));
    } else {
        vals = new T[n];
    }
}

__host__ __device__ void D_Array::Print() {
    printf("[ ");
#ifndef __CUDA_ARCH__
    if (isDevice) {
        printVector<<<1, 1>>>(*_device);
        hipDeviceSynchronize();
    } else
#endif
        printVectorBody(*this);
}

__host__ D_Array::~D_Array() {
    if (isDevice) {
        hipFree(vals);
        hipFree(_device);
    } else {
        delete[] vals;
    }
}
