#include "sparseDataStruct/helper/vector_helper.h"
#include "sparseDataStruct/vector_dense.hpp"

__host__ VectorDense::VectorDense(int n, bool isDevice)
    : n(n), isDevice(isDevice) {
    MemAlloc();
}

__host__ VectorDense::VectorDense(const VectorDense &m, bool copyToOtherMem)
    : VectorDense(n, m.isDevice ^ copyToOtherMem) {
    hipMemcpyKind memCpy =
        (m.isDevice)
            ? (isDevice) ? hipMemcpyDeviceToDevice : hipMemcpyDeviceToHost
            : (isDevice) ? hipMemcpyHostToDevice : hipMemcpyHostToHost;
    gpuErrchk(hipMemcpy(vals, m.vals, sizeof(T) * n, memCpy));

    gpuErrchk(hipMalloc(&_device, sizeof(VectorDense)));
    gpuErrchk(
        hipMemcpy(_device, this, sizeof(VectorDense), hipMemcpyHostToDevice));
}

__host__ void VectorDense::MemAlloc() {
    if (isDevice) {
        gpuErrchk(hipMalloc(&vals, n * sizeof(T)));
        gpuErrchk(hipMalloc(&_device, sizeof(VectorDense)));
        gpuErrchk(hipMemcpy(_device, this, sizeof(VectorDense),
                             hipMemcpyHostToDevice));
    } else {
        vals = new T[n];
    }
}

__host__ void VectorDense::Print() {
    printf("Vector values: ");
    if (isDevice) {
        printVector<<<1, 1>>>(_device);
        hipDeviceSynchronize();
    } else {
        printVectorBody(this);
    }
}
