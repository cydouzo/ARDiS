#include <assert.h>

#include "cudaHelper/cuda_error_check.h"
#include "cudaHelper/cusolverSP_error_check.h"
#include "cudaHelper/cusparse_error_check.h"
#include "cusparseOperations/row_ordering.hpp"
#include "sparseDataStruct/helper/matrix_helper.h"
#include <sparseDataStruct/matrix_element.hpp>
#include <sparseDataStruct/matrix_sparse.hpp>

__host__ MatrixSparse::MatrixSparse(int i_size, int j_size, int nnz,
                                    MatrixType type, bool isDevice)
    : nnz(nnz), i_size(i_size), j_size(j_size),
      isDevice(isDevice), type(type) {
    MemAlloc();
}

__host__ MatrixSparse::MatrixSparse(const MatrixSparse &m, bool copyToOtherMem)
    : MatrixSparse(m.i_size, m.j_size, m.nnz, m.type,
                   m.isDevice ^ copyToOtherMem) {
    loaded_elements = m.loaded_elements;
    assert(m.loaded_elements == m.nnz);
    hipMemcpyKind memCpy =
        (m.isDevice)
            ? (isDevice) ? hipMemcpyDeviceToDevice : hipMemcpyDeviceToHost
            : (isDevice) ? hipMemcpyHostToDevice : hipMemcpyHostToHost;
    gpuErrchk(hipMemcpy(vals, m.vals, sizeof(T) * nnz, memCpy));
    gpuErrchk(hipMemcpy(
        colPtr, m.colPtr,
        sizeof(int) * ((type == CSC) ? j_size + 1 : nnz), memCpy));
    gpuErrchk(hipMemcpy(
        rowPtr, m.rowPtr,
        sizeof(int) * ((type == CSR) ? i_size + 1 : nnz), memCpy));
}

__host__ void MatrixSparse::MemAlloc() {
    int rowPtrSize = (type == CSR) ? i_size + 1 : nnz;
    int colPtrSize = (type == CSC) ? j_size + 1 : nnz;
    if (isDevice) {
        gpuErrchk(hipMalloc(&vals, nnz * sizeof(T)));
        gpuErrchk(hipMalloc(&rowPtr, rowPtrSize * sizeof(int)));
        gpuErrchk(hipMalloc(&colPtr, colPtrSize * sizeof(int)));

        gpuErrchk(hipMalloc(&_device, sizeof(MatrixSparse)));
        gpuErrchk(hipMemcpy(_device, this, sizeof(MatrixSparse),
                             hipMemcpyHostToDevice));
    } else {
        vals = new T[nnz];
        rowPtr = new int[rowPtrSize];
        for (int i = 0; i < rowPtrSize; i++)
            rowPtr[i] = 0;
        colPtr = new int[colPtrSize];
        for (int i = 0; i < colPtrSize; i++)
            colPtr[i] = 0;
    }
}

__host__ __device__ void MatrixSparse::Print(int printCount) const {
#ifndef __CUDA_ARCH__
    if (isDevice) {
        printMatrix<<<1, 1>>>(_device, printCount);
        hipDeviceSynchronize();
    } else
#endif
        printMatrixBody(this, printCount);
}

__host__ __device__ void MatrixSparse::AddElement(int i, int j, T val) {
    assert(!isDevice);
    if (loaded_elements >= nnz) {
        printf("Error! The Sparse Matrix exceeded its moemory allocation!\n");
        return;
    }
    vals[loaded_elements] = val;
    if (type == CSC) {
        if (colPtr[j + 1] == 0)
            colPtr[j + 1] = colPtr[j];
        colPtr[j + 1]++;
    } else {
        colPtr[loaded_elements] = j;
    }
    if (type == CSR) {
        if (rowPtr[i + 1] == 0)
            rowPtr[i + 1] = rowPtr[i];
        rowPtr[i + 1]++;
    } else {
        rowPtr[loaded_elements] = i;
    }
    loaded_elements++;
}

// Get the value at index k of the sparse matrix
__host__ __device__ const T &MatrixSparse::Get(int k) const { return vals[k]; }
__host__ __device__ const T &MatrixSparse::GetLine(int i) const {
    if (type != CSR) {
        printf("Error! Doesn't work with other type than CSR");
    }
    return vals[rowPtr[i]];
}

__host__ __device__ T MatrixSparse::Lookup(int i, int j) const {
    for (MatrixElement elm(this); elm.HasNext(); elm.Next())
        if (elm.i == i && elm.j == j)
            return *elm.val;
    return 0;
}

__host__ void MatrixSparse::ToCompressedDataType(MatrixType toType,
                                                 bool orderBeforhand) {
    if (toType == COO) {
        if (IsConvertibleTo(CSR))
            toType = CSR;
        else if (IsConvertibleTo(CSC))
            toType = CSC;
        else {
            printf("Not convertible to any type!\n");
            return;
        }
    } else {
        assert(IsConvertibleTo(toType));
    }
    int newSize = (toType == CSR) ? i_size + 1 : j_size + 1;
    int *newArray;
    if (isDevice) {
        gpuErrchk(hipMalloc(&newArray, newSize * sizeof(int)));
        convertArray<<<1, 1>>>((toType == CSR) ? rowPtr : colPtr, nnz,
                               newArray, newSize);
        hipFree((toType == CSR) ? rowPtr : colPtr);
    } else {
        newArray = new int[newSize];
        convertArrayBody((toType == CSR) ? rowPtr : colPtr, nnz,
                         newArray, newSize);
        if (toType == CSR)
            delete[] rowPtr;
        else
            delete[] colPtr;
    }
    if (toType == CSR)
        rowPtr = newArray;
    else
        colPtr = newArray;
    type = toType;
    if (isDevice) {
        gpuErrchk(hipMemcpy(_device, this, sizeof(MatrixSparse),
                             hipMemcpyHostToDevice));
        gpuErrchk(hipDeviceSynchronize());
    }
}

__host__ bool MatrixSparse::IsConvertibleTo(MatrixType toType) const {
    assert(toType != type);
    if (toType == COO)
        return true;
    if (type != COO)
        return false;
    int *analyzedArray = (toType == CSR) ? rowPtr : colPtr;
    bool isOK = true;
    if (isDevice) {
        bool *_isOK;
        gpuErrchk(hipMalloc(&_isOK, sizeof(bool)));
        checkOrdered<<<1, 1>>>(analyzedArray, nnz, _isOK);
        gpuErrchk(
            hipMemcpy(&isOK, _isOK, sizeof(bool), hipMemcpyDeviceToHost));
        gpuErrchk(hipFree(_isOK));
        gpuErrchk(hipDeviceSynchronize());
    } else {
        checkOrderedBody(analyzedArray, nnz, &isOK);
    }
    return isOK;
}

__host__ void MatrixSparse::ConvertMatrixToCSR() {
    if (type == CSR)
        throw("Error! Already CSR type \n");
    if (type == CSC)
        throw("Error! Already CSC type \n");
    if (!IsConvertibleTo(CSR)) {
        RowOrdering(*this);
    }
    assert(IsConvertibleTo(CSR));
    ToCompressedDataType(CSR);
    assert(type == CSR);
}

__host__ void MatrixSparse::MakeDescriptor() {
    if (descr == NULL) {
        cusparseErrchk(hipsparseCreateMatDescr(&descr));
        hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    } else {
        printf("Matrix already has a descriptor!");
    }
}

__host__ bool MatrixSparse::IsSymetric() {
    bool *_return = new bool;
    if (isDevice) {
        bool *_returnGpu;
        gpuErrchk(hipMalloc(&_returnGpu, sizeof(bool)));
        IsSymetricKernel<<<1, 1>>>(_device, _returnGpu);
        hipDeviceSynchronize();
        gpuErrchk(hipMemcpy(_return, _returnGpu, sizeof(bool),
                             hipMemcpyDeviceToHost));
        gpuErrchk(hipFree(_returnGpu));
        gpuErrchk(hipDeviceSynchronize());
    } else {
        IsSymetricBody(this, _return);
    }
    return *_return;
}

typedef hipsparseStatus_t (*FuncSpar)(...);
__host__ void MatrixSparse::OperationCuSparse(void *function,
                                              hipsparseHandle_t &handle,
                                              bool addValues, void *pointer1,
                                              void *pointer2) {
    if (addValues) {
        printf("This function is not complete");
    } else {
        if (pointer1)
            if (pointer2) {
                cusparseErrchk(((FuncSpar)function)(
                    handle, i_size, j_size, loaded_elements, rowPtr, colPtr,
                    pointer1, pointer2));
            } else {
                cusparseErrchk(((FuncSpar)function)(handle, i_size, j_size,
                                                    loaded_elements, rowPtr,
                                                    colPtr, pointer1));
            }
        else
            printf("This function is not complete");
    }
}

typedef hipsolverStatus_t (*FuncSolv)(...);
__host__ void MatrixSparse::OperationCuSolver(void *function,
                                              hipsolverSpHandle_t &handle, T *b,
                                              T *xOut, int *singularOut) {
    cusolverErrchk(((FuncSolv)function)(handle, i_size, loaded_elements, descr,
                                        vals, rowPtr, colPtr, b, 0.0, 0, xOut,
                                        singularOut));
    // TODO : SymOptimization
}

__host__ MatrixSparse::~MatrixSparse() {
    if (isDevice) {
        gpuErrchk(hipFree(vals));
        gpuErrchk(hipFree(rowPtr));
        gpuErrchk(hipFree(colPtr));
        gpuErrchk(hipFree(_device));
    } else {
        delete[] vals;
        delete[] rowPtr;
        delete[] colPtr;
    }
}
