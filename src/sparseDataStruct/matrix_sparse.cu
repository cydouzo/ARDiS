#include <assert.h>

#include "cudaHelper/cuda_error_check.h"
#include "cudaHelper/cusolverSP_error_check copy.h"
#include "cudaHelper/cusparse_error_check.h"
#include "sparseDataStruct/helper/matrix_helper.h"
#include <sparseDataStruct/matrix_element.hpp>
#include <sparseDataStruct/matrix_sparse.hpp>

__host__ MatrixSparse::MatrixSparse(int i_size, int j_size, int n_elements,
                                    MatrixType type, bool isDevice)
    : n_elements(n_elements), i_size(i_size), j_size(j_size),
      isDevice(isDevice), type(type) {
    MemAlloc();
}

__host__ MatrixSparse::MatrixSparse(const MatrixSparse &m, bool copyToOtherMem)
    : MatrixSparse(m.i_size, m.j_size, m.n_elements, m.type,
                   m.isDevice ^ copyToOtherMem) {
    hipMemcpyKind memCpy =
        (m.isDevice)
            ? (isDevice) ? hipMemcpyDeviceToDevice : hipMemcpyDeviceToHost
            : (isDevice) ? hipMemcpyHostToDevice : hipMemcpyHostToHost;
    gpuErrchk(hipMemcpy(vals, m.vals, sizeof(T) * n_elements, memCpy));
    gpuErrchk(hipMemcpy(
        colPtr, m.colPtr,
        sizeof(int) * ((type == CSC) ? j_size + 1 : n_elements), memCpy));
    gpuErrchk(hipMemcpy(
        rowPtr, m.rowPtr,
        sizeof(int) * ((type == CSR) ? i_size + 1 : n_elements), memCpy));
}

__host__ void MatrixSparse::MemAlloc() {
    int rowPtrSize = (type == CSR) ? i_size + 1 : n_elements;
    int colPtrSize = (type == CSC) ? j_size + 1 : n_elements;
    if (isDevice) {
        gpuErrchk(hipMalloc(&vals, n_elements * sizeof(T)));
        gpuErrchk(hipMalloc(&rowPtr, rowPtrSize * sizeof(int)));
        gpuErrchk(hipMalloc(&colPtr, colPtrSize * sizeof(int)));

        gpuErrchk(hipMalloc(&_device, sizeof(MatrixSparse)));
        gpuErrchk(hipMemcpy(_device, this, sizeof(MatrixSparse),
                             hipMemcpyHostToDevice));
    } else {
        vals = new T[n_elements];
        rowPtr = new int[rowPtrSize];
        for (int i = 0; i < rowPtrSize; i++)
            rowPtr[i] = 0;
        colPtr = new int[colPtrSize];
        for (int i = 0; i < colPtrSize; i++)
            colPtr[i] = 0;
    }
}

__host__ void MatrixSparse::Print() const {
    if (isDevice) {
        printMatrix<<<1, 1>>>(_device);
        hipDeviceSynchronize();
    } else
        printMatrixBody(this);
}

__host__ __device__ void MatrixSparse::AddElement(int k, int i, int j,
                                                  const T val) {
    assert(!isDevice);
    vals[k] = val;
    if (type == CSC) {
        if (colPtr[j + 1] == 0)
            colPtr[j + 1] = colPtr[j];
        colPtr[j + 1]++;
    } else {
        colPtr[k] = j;
    }
    if (type == CSR) {
        if (rowPtr[i + 1] == 0)
            rowPtr[i + 1] = rowPtr[i];
        rowPtr[i + 1]++;
    } else {
        rowPtr[k] = i;
    }
}

// Get the value at index k of the sparse matrix
__host__ __device__ const T &MatrixSparse::Get(int k) const { return vals[k]; }

__host__ void MatrixSparse::ToCompressedDataType(MatrixType toType,
                                                 bool orderBeforhand) {
    if (toType == COO) {
        if (IsConvertibleTo(CSR))
            toType = CSR;
        else if (IsConvertibleTo(CSC))
            toType = CSC;
        else {
            printf("Not convertible to any type!\n");
            return;
        }
    } else {
        assert(IsConvertibleTo(toType));
    }
    int newSize = (toType == CSR) ? i_size + 1 : j_size + 1;
    int *newArray;
    if (isDevice) {
        gpuErrchk(hipMalloc(&newArray, newSize * sizeof(int)));
        gpuErrchk(hipDeviceSynchronize());
        convertArray<<<1, 1>>>((toType == CSR) ? rowPtr : colPtr, n_elements,
                               newArray, newSize);
        gpuErrchk(hipDeviceSynchronize());
    } else {
        newArray = new int[newSize];
        convertArrayBody((toType == CSR) ? rowPtr : colPtr, n_elements,
                         newArray, newSize);
    }
    if (toType == CSR)
        rowPtr = newArray;
    else
        colPtr = newArray;
    type = toType;
    if (isDevice)
        gpuErrchk(hipMemcpy(_device, this, sizeof(MatrixSparse),
                             hipMemcpyHostToDevice));
}

__host__ bool MatrixSparse::IsConvertibleTo(MatrixType toType) const {
    assert(toType != type);
    if (toType == COO)
        return true;
    if (type != COO)
        return false;
    int *analyzedArray = (toType == CSR) ? rowPtr : colPtr;
    bool isOK = true;
    if (isDevice) {
        bool *_isOK;
        gpuErrchk(hipMalloc(&_isOK, sizeof(bool)));
        checkOrdered<<<1, 1>>>(analyzedArray, n_elements, _isOK);
        gpuErrchk(
            hipMemcpy(&isOK, _isOK, sizeof(bool), hipMemcpyDeviceToHost));
    } else {
        for (int k = 1; k < n_elements && isOK; k++) {
            isOK = isOK && analyzedArray[k] >= analyzedArray[k - 1];
        }
    }
    return isOK;
}

__host__ void MatrixSparse::MakeDescriptor() {
    if (descr == NULL) {
        cusparseErrchk(hipsparseCreateMatDescr(&descr));
        hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    } else {
        printf("Matrix already has a descriptor!");
    }
}

typedef hipsparseStatus_t (*FuncSpar)(...);
__host__ void MatrixSparse::OperationCuSparse(void *function,
                                              hipsparseHandle_t &handle,
                                              bool addValues, void *pointer1,
                                              void *pointer2) {
    if (addValues) {
        printf("This function is not complete");
    } else {
        if (pointer1)
            if (pointer2) {
                cusparseErrchk(((FuncSpar)function)(handle, i_size, j_size,
                                                    n_elements, rowPtr, colPtr,
                                                    pointer1, pointer2));
            } else {
                cusparseErrchk(((FuncSpar)function)(handle, i_size, j_size,
                                                    n_elements, rowPtr, colPtr,
                                                    pointer1));
            }
        else
            printf("This function is not complete");
    }
}

typedef hipsolverStatus_t (*FuncSolv)(...);
__host__ void MatrixSparse::OperationCuSolver(void *function,
                                              hipsolverSpHandle_t &handle, T *b,
                                              T *xOut, int *singularOut) {
    cusolverErrchk(((FuncSolv)function)(handle, i_size, n_elements, descr, vals,
                                        rowPtr, colPtr, b, 0.001, 0, xOut,
                                        singularOut));
    // TODO : SymOptimization
}

__host__ MatrixSparse::~MatrixSparse() {
    if (isDevice) {
        gpuErrchk(hipFree(vals));
        gpuErrchk(hipFree(rowPtr));
        gpuErrchk(hipFree(colPtr));
        gpuErrchk(hipFree(_device));
    } else {
        delete[] vals;
        delete[] rowPtr;
        delete[] colPtr;
    }
}
