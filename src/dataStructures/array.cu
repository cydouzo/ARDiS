#include "hip/hip_runtime.h"
#include "dataStructures/array.hpp"
#include "dataStructures/hd_data.hpp"
#include "dataStructures/helper/vector_helper.h"
#include "hediHelper/cuda/cuda_thread_manager.hpp"
#include "helper/apply_operation.h"
#include "matrixOperations/basic_operations.hpp"

__host__ D_Array::D_Array(int n, bool isDevice) : n(n), isDevice(isDevice) {
    MemAlloc();
}

__host__ D_Array::D_Array(const D_Array &m, bool copyToOtherMem)
    : D_Array(m.n, m.isDevice ^ copyToOtherMem) {
    hipMemcpyKind memCpy =
        (m.isDevice)
            ? (isDevice) ? hipMemcpyDeviceToDevice : hipMemcpyDeviceToHost
            : (isDevice) ? hipMemcpyHostToDevice : hipMemcpyHostToHost;
    gpuErrchk(hipMemcpy(vals, m.vals, sizeof(T) * n, memCpy));
}

__host__ void D_Array::operator=(const D_Array &other) {
    assert(isDevice == other.isDevice);
    MemFree();
    n = other.n;
    MemAlloc();
    hipMemcpyKind memCpy =
        (other.isDevice)
            ? (isDevice) ? hipMemcpyDeviceToDevice : hipMemcpyDeviceToHost
            : (isDevice) ? hipMemcpyHostToDevice : hipMemcpyHostToHost;
    gpuErrchk(hipMemcpy(vals, other.vals, sizeof(T) * n, memCpy));
}

// __host__ void D_Array::Swap(D_Array &other) {
//     assert(isDevice == other.isDevice);
//     n = other.n;
//     std::swap(_device, other._device);
//     std::swap(vals, other.vals);
//     other.MemFree();
// }

__host__ void D_Array::Resize(int n) {
    MemFree();
    this->n = n;
    MemAlloc();
}

__host__ hipsparseDnVecDescr_t D_Array::MakeDescriptor() {
    hipsparseDnVecDescr_t descr;
    cusparseErrchk(hipsparseCreateDnVec(&descr, n, vals, T_Cuda));
    return descr;
}

__host__ T D_Array::Norm() {
    assert(isDevice);
    HDData<T> norm;
    Dot(*this, *this, norm(true));
    norm.SetHost();
    return norm();
}

__host__ void D_Array::Fill(T value) {
    auto setTo = [value] __device__(T & a) { a = value; };
    ApplyFunction(*this, setTo);
}

__host__ void D_Array::Prune(T value) {
    auto setTo = [value] __device__(T & a) {
        if (a < value)
            a = value;
    };
    ApplyFunction(*this, setTo);
}
__host__ void D_Array::PruneUnder(T value) {
    auto setTo = [value] __device__(T & a) {
        if (a > value)
            a = value;
    };
    ApplyFunction(*this, setTo);
}

__host__ __device__ void D_Array::Print(int printCount) {
    printf("[ ");
#ifndef __CUDA_ARCH__
    if (isDevice) {
        printVector<<<1, 1>>>(*_device, printCount);
        gpuErrchk(hipDeviceSynchronize());
    } else
#endif
        printVectorBody(*this, printCount);
}

__host__ D_Array::~D_Array() { MemFree(); }

__host__ void D_Array::MemAlloc() {
    if (n > 0)
        if (isDevice) {
            gpuErrchk(hipMalloc(&vals, n * sizeof(T)));
            gpuErrchk(hipMalloc(&_device, sizeof(D_Array)));
            gpuErrchk(hipMemcpy(_device, this, sizeof(D_Array),
                                 hipMemcpyHostToDevice));
        } else {
            vals = new T[n];
        }
}

__host__ void D_Array::MemFree() {
    if (n > 0)
        if (isDevice) {
            gpuErrchk(hipFree(vals));
            gpuErrchk(hipFree(_device));
            gpuErrchk(hipDeviceSynchronize());
        } else {
            delete[] vals;
        }
}