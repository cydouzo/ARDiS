#include "dataStructures/array.hpp"
#include "dataStructures/hd_data.hpp"
#include "dataStructures/helper/vector_helper.h"
#include "helper/apply_operation.h"
#include "helper/cuda/cuda_thread_manager.hpp"
#include "matrixOperations/basic_operations.hpp"
#include "sstream"

__device__ __host__ void CallError(AccessError error) {
    switch (error) {
    case AccessDeviceOnHost:
        printf("Error, trying to access device array from the host");
    case AccessHostOnDevice:
        printf("Error, trying to access host array from the device");
    }
}

template <typename C>
__host__ D_Array<C>::D_Array(int n, bool isDevice) : n(n), isDevice(isDevice) {
    MemAlloc();
}

template <typename C>
__host__ D_Array<C>::D_Array(const D_Array<C> &m, bool copyToOtherMem)
    : D_Array<C>(m.n, m.isDevice ^ copyToOtherMem) {
    hipMemcpyKind memCpy =
        (m.isDevice)
            ? (isDevice) ? hipMemcpyDeviceToDevice : hipMemcpyDeviceToHost
            : (isDevice) ? hipMemcpyHostToDevice : hipMemcpyHostToHost;
    gpuErrchk(hipMemcpy(data, m.data, sizeof(C) * n, memCpy));
}

template <typename C>
__host__ D_Array<C>::D_Array(D_Array<C> &&other) : D_Array(0, other.isDevice) {
    *this = other;
}

template <typename C>
__host__ void D_Array<C>::operator=(const D_Array<C> &other) {
    if (isDevice != other.isDevice)
        if (isDevice)
            throw("You cannot move an array host array into a device array");
        else
            throw("You cannot move an array device array into a host array");
    MemFree();
    n = other.n;
    n_dataholders = other.n_dataholders;
    *n_dataholders += 1;
    data = other.data;
    if (isDevice)
        _device = other._device;
}

template <typename C> __host__ void D_Array<C>::Resize(int n) {
    MemFree();
    this->n = n;
    MemAlloc();
}

template <typename C> __host__ D_Array<C>::~D_Array<C>() { MemFree(); }

template <typename C> __host__ void D_Array<C>::MemAlloc() {
    if (n > 0) {
        n_dataholders = new int[1];
        *n_dataholders = 1;
        if (isDevice) {
            gpuErrchk(hipMalloc(&data, n * sizeof(T)));
            gpuErrchk(hipMalloc(&_device, sizeof(D_Array<C>)));
            gpuErrchk(hipMemcpy(_device, this, sizeof(D_Array<C>),
                                 hipMemcpyHostToDevice));
        } else {
            data = new C[n];
        }
    }
}

template <typename C> __host__ void D_Array<C>::MemFree() {
    if (n > 0) {
        *n_dataholders -= 1;
        if (*n_dataholders == 0) {
            if (isDevice) {
                gpuErrchk(hipFree(data));
                gpuErrchk(hipFree(_device));
                gpuErrchk(hipDeviceSynchronize());
            } else {
                delete[] data;
            }
        }
    }
}

template <typename C>
__host__ __device__ void D_Array<C>::Print(int printCount) const {
#ifndef __CUDA_ARCH__
    if (isDevice) {
        gpuErrchk(hipDeviceSynchronize());
        printVectorK<<<1, 1>>>(*_device, printCount);
        gpuErrchk(hipDeviceSynchronize());
    } else
#else
    if (!isDevice)
        CallError(AccessHostOnDevice);
    else
#endif
        printVectorBody(*this, printCount);
}

template <typename C> __host__ __device__ C &D_Array<C>::at(int i) {
#ifndef __CUDA_ARCH__
    if (isDevice)
        CallError(AccessDeviceOnHost);
#else

    if (!isDevice)
        CallError(AccessHostOnDevice);
#endif
    return data[i];
}

template <typename C> __host__ __device__ int D_Array<C>::size() { return n; }
template <typename C> __host__ __device__ bool D_Array<C>::IsDevice() {
    return isDevice;
}

template <typename C>
__host__ hipsparseDnVecDescr_t D_Array<C>::MakeDescriptor() {
    hipsparseDnVecDescr_t descr;
    cusparseErrchk(hipsparseCreateDnVec(&descr, n, data, T_Cuda));
    return descr;
}

template <typename C> __host__ void D_Array<C>::Fill(C value) {
    auto setTo = [value] __device__(C & a) { a = value; };
    ApplyFunction(*this, setTo);
}

#define quote(x) #x

__host__ void D_Vector::Prune(T value) {
    auto setTo = [value] __device__(T & a) {
        if (a < value)
            a = value;
    };
    ApplyFunction(*this, setTo);
}
__host__ void D_Vector::PruneUnder(T value) {
    auto setTo = [value] __device__(T & a) {
        if (a > value)
            a = value;
    };
    ApplyFunction(*this, setTo);
}

__host__ std::string D_Vector::ToString() {
    int printCount = 5;
    std::stringstream strs;
    strs << "[ ";
    T *printBuffer = new T[printCount + 1];
    hipMemcpy(printBuffer, data, sizeof(T) * printCount,
               (isDevice) ? hipMemcpyDeviceToHost : hipMemcpyHostToHost);
    hipMemcpy(printBuffer + printCount, data + n - 1, sizeof(T),
               (isDevice) ? hipMemcpyDeviceToHost : hipMemcpyHostToHost);

    for (int i = 0; i < (n - 1) && i < printCount; i++)
        strs << printBuffer[i] << ", ";
    if (printCount < n - 1)
        strs << "... ";
    strs << printBuffer[printCount] << "]";
    delete[] printBuffer;
    return strs.str();
}
